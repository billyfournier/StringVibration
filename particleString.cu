#include "hip/hip_runtime.h"
//nvcc particleString.cu -o temp -lglut -lGL -lm
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>

#define PI 3.14159265359 

#define X_WINDOW 1000
#define Y_WINDOW 700

#define L 0.67945       //Length of string in Meters
#define D 0.18415       //Laterail displacement of string in Meters
#define MASS  0.006     //Mass of string in Kilograms
#define K 38.6220      	//Spring constant of string in Newton/Meters
#define Tention 1.0 //5.3955 	//Resting tention in Newtons
#define N 1002          //number of bodies
#define P 1002	        //number of bodies per block
#define DAMP 0.00005   	//Air resistance

#define X_MAX (L/2.0)
#define X_MIN -(L/2.0)
#define X_SCALE 0.1

#define Y_MAX D
#define Y_MIN -D
#define Y_SCALE 0.1

#define TIME_DURATION	1000.0
#define STEP_SIZE        0.0000005
#define TIME_STEP_BETWEEN_VIEWING 100 
float *X_CPU, *Y_CPU, *VX_CPU, *VY_CPU, *AX_CPU, *AY_CPU;  //CPU pointers

float *X_GPU, *Y_GPU, *VX_GPU, *VY_GPU, *AX_GPU, *AY_GPU; //GPU pointers

dim3 dimBlock, dimGrid; //Block and Grid Dimensions


void 	SetUpCudaDevices() // Sets up the architecture for processes
	{	
		//Threads in a block
		dimBlock.x = P;
		dimBlock.y = 1;
		dimBlock.z = 1;
	
		//Blocks in a grid
		dimGrid.x = 1;
		dimGrid.y = 1;
		dimGrid.z = 1;
	}

void 	AllocateMemory()
{					
	//Allocate Device (GPU) Memory, & allocates the value of the specific pointer/array
	hipMalloc(&X_GPU, N*sizeof(float));
	hipMalloc(&Y_GPU, N*sizeof(float));
	hipMalloc(&VX_GPU,N*sizeof(float));
	hipMalloc(&VY_GPU,N*sizeof(float));
	hipMalloc(&AX_GPU,N*sizeof(float));
	hipMalloc(&AY_GPU,N*sizeof(float));

	//Allocate Host (CPU) Memory
	X_CPU  = (float*)malloc(N*sizeof(float)); //(float*) to prevent from being a void
	Y_CPU  = (float*)malloc(N*sizeof(float));
	VX_CPU = (float*)malloc(N*sizeof(float));
	VY_CPU = (float*)malloc(N*sizeof(float));
	AX_CPU = (float*)malloc(N*sizeof(float));
	AY_CPU = (float*)malloc(N*sizeof(float));
}

float x_machine_to_x_screen(int x)
{
	return( (2.0*x)/X_WINDOW-1.0 );
}

float y_machine_to_y_screen(int y)
{
	return( -(2.0*y)/Y_WINDOW+1.0 );
}

/*	Takes machine x and y which start in the upper left corner and go from zero to X_WINDOW
	left to right and form zero to Y_WINDOW top to bottom and transslates this into world 
	points which are a X_MIN to X_MAX, Y_MIN to Y_MAX window.
*/
float x_machine_to_x_world(int x)
{
	float range;
	range = X_MAX - X_MIN;
	return( (range/X_WINDOW)*x + X_MIN );
}

float y_machine_to_y_world(int y)
{
	float range;
	range = Y_MAX - Y_MIN;
	return(-((range/Y_WINDOW)*y - Y_MAX));
}

/*	Take world  points to screen points 
*/
float x_world_to_x_screen(float x)
{
	float range;
	range = X_MAX - X_MIN;
	return( -1.0 + 2.0*(x - X_MIN)/range );
}

float y_world_to_y_screen(float y)
{
	float range;
	range = Y_MAX - Y_MIN;
	return( -1.0 + 2.0*(y - Y_MIN)/range );
}

void 	draw_spring(float *x, float *y)
	{
		int i;

		glPointSize(1.0);
		glClear(GL_COLOR_BUFFER_BIT);

		glColor3f(1.0,1.0,0.0);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(x[0]),y_world_to_y_screen(y[0]));
		glEnd();

		glColor3f(1.0,1.0,1.0);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(x[N-1]),y_world_to_y_screen(y[N-1]));
		glEnd();
			
		glColor3f(1.0,0.0,0.0);
		for(i = 1; i < N-1; i++)
		{			
			glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(x[i]),y_world_to_y_screen(y[i]));
			glEnd();
			
		}
		glFlush();
	}

__global__ void Findforce(float *x, float *y, float *vx, float *vy, float *ax, float *ay, float l, float mass, float k) // This is the kernel, it is the function that is being fed to the GPU.
{	
	int id = threadIdx.x;
	float dx, dy, d2, d, f;

	if(0 < threadIdx.x && threadIdx.x < P-1)
	{
		dx = x[id-1]-x[id];
		dy = y[id-1]-y[id];
		d2 = dx*dx + dy*dy;
		d  = sqrt(d2);
		f = k*(d-l) + Tention;

		ax[id] += (f*dx/d)/mass;
		ay[id] += (f*dy/d)/mass;
			
			
		dx = x[id+1]-x[id];
		dy = y[id+1]-y[id];
		d2 = dx*dx + dy*dy;
		d  = sqrt(d2);
		f = k*(d-l) + Tention;		
		ax[id] += (f*dx/d)/mass;
		ay[id] += (f*dy/d)/mass;
		
		ax[id] += (-DAMP*vx[id])/mass;
		ay[id] += (-DAMP*vy[id])/mass;
	}
	__syncthreads();
}

void n_body()
{
	float l, time, dt;
	int draw_count,i;
	float particleMass, particleK;

	SetUpCudaDevices();
	AllocateMemory();

	l = L/(N-1);

	time = 0.0;
	dt = STEP_SIZE;
	draw_count = 0;
	
	particleMass = (MASS/L)/(float)N;
	particleK = K*(float(N-1));
	//particleK = K;

	X_CPU[0] = -L/2.0;
	Y_CPU[0] = 0.0;
	X_CPU[N-1] = L/2.0;
	Y_CPU[N-1] = 0.0;
	
	for(i=1; i<(N-1); i++)
	{
		X_CPU[i] = -L/2.0 + l*(i);
		if(X_CPU[i] <= 0.0) Y_CPU[i] = (2.0*D/L)*(X_CPU[i]+L/2.0);
		if(X_CPU[i] >  0.0) Y_CPU[i] = D - (2.0*D/L)*(X_CPU[i]);
		VX_CPU[i] = 0.0;
		VY_CPU[i] = 0.0;
	}
	draw_spring(X_CPU, Y_CPU);
	
	printf("\ninter a character in the terminal\n");
	getchar();
	
	while(time < TIME_DURATION)
	{
		for(i=0; i<N; i++) 
		{
			AX_CPU[i] = 0.0;
			AY_CPU[i] = 0.0;
		}	
			
		//Copy Memory from CPU to GPU		
		hipMemcpyAsync(X_GPU,   X_CPU, N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpyAsync(Y_GPU,   Y_CPU, N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpyAsync(VX_GPU, VX_CPU, N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpyAsync(VY_GPU, VY_CPU, N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpyAsync(AX_GPU, AX_CPU, N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpyAsync(AY_GPU, AY_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	
		//Launch Kernel	
		Findforce<<<dimGrid, dimBlock>>>(X_GPU, Y_GPU, VX_GPU, VY_GPU, AX_GPU, AY_GPU, l, particleMass, particleK);
	
		//Copy Memory from GPU to CPU	
		hipMemcpyAsync(AY_CPU, AY_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpyAsync(AX_GPU, AX_CPU, N*sizeof(float), hipMemcpyDeviceToHost);
	
		for(i=1; i < (N-1); i++)
		{
			VX_CPU[i] += AX_CPU[i]*dt;
			VY_CPU[i] += AY_CPU[i]*dt;
			X_CPU[i]  += VX_CPU[i]*dt;
			Y_CPU[i]  += VY_CPU[i]*dt;
		}

		if(draw_count == TIME_STEP_BETWEEN_VIEWING)
		{
			draw_spring(X_CPU, Y_CPU);
			draw_count = 0;
		}

		time = time + dt;
		draw_count++;
	}
}

void display()
{
	//glClear(GL_COLOR_BUFFER_BIT);
	//glFlush();
	
	n_body();
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitWindowSize(X_WINDOW,Y_WINDOW);
	glutInitWindowPosition(0,0);
	glutCreateWindow("BOX");
	glutDisplayFunc(display);
	glutMainLoop();
}




